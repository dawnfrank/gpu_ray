#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

extern "C" hipError_t InitCuda(int w,int h,unsigned char** dev_bitmap);
extern "C" hipError_t CalculateCuda(int w, int h, unsigned char* dev_bitmap,unsigned char* host_bitmap);
extern "C" void DeinitCuda(unsigned char* dev_bitmap);

__global__ void RayKernel(int w, int h,unsigned char* dev_bitmap)
{
	int i = blockIdx.x;
	int j = blockIdx.y;
	dev_bitmap[i * 4] = int(255.99*double(i)/double(w));
	dev_bitmap[i * 4 + 1] = int(255.99*double(j) / double(w));
	dev_bitmap[i * 4 + 2] = int(255.99*0.2);
	dev_bitmap[i * 4 + 3] = 1;
}

hipError_t CalculateCuda(int w, int h, unsigned char* dev_bitmap, unsigned char* host_bitmap) {
	hipError_t cudaStatus;
	int image_size = w * h * 4;

	// Launch a kernel on the GPU with one thread for each element.
	dim3 grid(w, h);
	RayKernel << <grid, 1 >> >(w,h,dev_bitmap);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(host_bitmap, dev_bitmap, image_size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}
	return cudaStatus;
}

hipError_t InitCuda(int w, int h, unsigned char** dev_bitmap) {
	hipError_t cudaStatus;
	int image_size = w * h * 4;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	// Allocate GPU buffers for three vectors (two input, one output)   
	cudaStatus = hipMalloc((void**)dev_bitmap, image_size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	return cudaStatus;
}

void DeinitCuda(unsigned char* dev_bitmap) {
	hipFree(dev_bitmap);
}
